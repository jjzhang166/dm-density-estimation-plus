#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>

using namespace std;

#include "types.h"
#include "kernel.h"
#include "tetrahedron.h"
#include "tetrastream.h"
#include "gridmanager.h"

#include "tetrahedron.cu"

using namespace std;

namespace kernel_space{
	int sub_grid_size_;
	int num_tetra_ = 0;

	Tetrahedron * dev_tetras;						//the tetrahedrons in the GPU memory
	Tetrahedron * tetras_v;							//the tetrahedrons in the CPU memory
	GridManager * gridmanager;						//grid manager
	TetraStream * tetrastream;						//tetrahedron stream

	REAL * dev_grids;								//the grids in the GPU memory
	Point * dev_grid_velocity;						//velocity grids
	int * dev_tetra_mem;							//each element specifies the total tetras a block have
	int * dev_tetra_select;							//tetra hedron selected in this list

	long TETRA_LIST_MEM_LIM = 128*1024*1024;		//128 for the memory lists
	int current_tetra_list_ind = 0;					//the current grid block, which is already calculated tetrahedron selection 

	int * tetramem;
	int * tetramem_list;							//the tetramemory list

	GridVelocityManager * grid_velocity = NULL;
	bool is_Velocity = false;
}

using namespace kernel_space;

__global__ void tetraSplatter(Tetrahedron * dtetra, int ntetra, REAL * dgrids,
	int gsize, int sub_gsize, 
	int * tetra_mem, int * tetra_selection, int sub_ind,  int numsubgridsize,
	REAL box = 32000, REAL x0 = 0, REAL y0 = 0, REAL z0 = 0, bool isvelocity = false, 
	Point * dev_grid_velocity = NULL){

	int loop_i = 0;
	int i, j, k;
	double d0=0.0, d1=0.0, d2=0.0, d3=0.0, d4=0.0;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;

	if(i >= sub_gsize)
		return;
	if(j >= sub_gsize)
		return;
	if(k >= sub_gsize)
		return;

	int startind = 0;
	if(sub_ind > 0){
		startind = tetra_mem[sub_ind - 1];
	}
	int endind = tetra_mem[sub_ind];

    REAL dx2 = box/gsize/2.0f;

	//ntetra
	//for(loop_i = 0; loop_i < ntetra; loop_i ++){
	for(loop_i = startind; loop_i < endind; loop_i ++){
		Tetrahedron * tetra = &dtetra[tetra_selection[loop_i]];
		
		REAL ng = gsize;
		int sgs = sub_gsize;

		//calculate the actual coordinate
		Point p;//getPoint(i, j, k);
		p.x = i / (REAL) ng * box + x0 + dx2; 
		p.y = j / (REAL) ng * box + y0 + dx2;
		p.z = k / (REAL) ng * box + z0 + dx2;
		

		if(tetra->isInTetra(p, d0, d1, d2, d3, d4)){
			//testing
			
			//REAL ivol = 1.0 / tetra->volume;
			dgrids[i + j * sgs + k * sgs * sgs] += tetra->invVolume;

			if(isvelocity){
				//interpolate on the velocity
				Point vel;
				vel.x = (d1/d0 * tetra->velocity1.x + d2/d0 * tetra->velocity2.x + 
						d3/d0 * tetra->velocity3.x + d4/d0 * tetra->velocity4.x) * tetra->invVolume;

				vel.y = (d1/d0 * tetra->velocity1.y + d2/d0 * tetra->velocity2.y + 
						d3/d0 * tetra->velocity3.y + d4/d0 * tetra->velocity4.y) * tetra->invVolume;

				vel.z = (d1/d0 * tetra->velocity1.z + d2/d0 * tetra->velocity2.z + 
						d3/d0 * tetra->velocity3.z + d4/d0 * tetra->velocity4.z) * tetra->invVolume;
				dev_grid_velocity[i + j * sgs + k * sgs * sgs].x += vel.x;
				dev_grid_velocity[i + j * sgs + k * sgs * sgs].y += vel.y;
				dev_grid_velocity[i + j * sgs + k * sgs * sgs].z += vel.z;
			}
		}
	}
}

// numsubgrid is the gridsize / subgridsize
// get a actual coordinate of the i, j, k
__device__ Point getPoint(int ind, int i, int j, int k, int subgridsize, 
		int gridsize, int numsubgrid, float box){
	int ai, aj, ak;

	int i0 = (ind % numsubgrid) * subgridsize;
	int j0 = (ind / numsubgrid % numsubgrid) * subgridsize;
	int k0 = (ind / numsubgrid / numsubgrid % numsubgrid) * subgridsize;

	ai = i + i0;
	aj = j + j0;
	ak = k + k0;

	float fx, fy, fz;

	Point retP;
	fx = (float) ai / (float) gridsize;
	fy = (float) aj / (float) gridsize;
	fz = (float) ak / (float) gridsize;

	retP.x = fx * box;
	retP.y = fy * box;
	retP.z = fz * box;
	return retP;
}

// nsg = gs / subs
// vox_vel = box^3/ng^3;
//check whether the tetrahedron cuboid is in touch with the grid sub-block
//v1 and v8 are the two diagnal point of the cube
__device__ bool isInTouch(int ind, int subgs, int gs, int nsg, float box, float dx2, 
	Tetrahedron * tetra, Point &v1, Point &v8){
	//moved
	/*
	Point v1, v8;
	v1 = getPoint(ind, 0, 0, 0,subgs, gs, nsg, box);
	v8 = getPoint(ind, subgs,subgs,subgs, subgs, gs, nsg, box);
	*/

	REAL minx = tetra->minx();
	REAL maxx = tetra->maxx();
	REAL miny = tetra->miny();
	REAL maxy = tetra->maxy();
	REAL minz = tetra->minz();
	REAL maxz = tetra->maxz();

	/* moved to the tetrahedron reading
	if(maxx - minx > box / 2.0)
			return false;

	if(maxy - miny > box / 2.0)
			return false;

	if(maxz - minz > box / 2.0)
			return false;
	*/

	if (minx > v8.x + dx2 || maxx < v1.x - dx2
		|| miny > v8.y + dx2 || maxy < v1.y - dx2
		|| minz > v8.z + dx2 || maxz < v1.z - dx2){
		return false;
	}
	//test
	/*if (minx > v8.x|| maxx < v1.x
		|| miny > v8.y || maxy < v1.y
		|| minz > v8.z || maxz < v1.z){
		return false;
	}*/

	return true;

}


//compute how many tetrahedrons are in touch with a certain subblock of the density grid
__global__ void computeTetraMem(Tetrahedron * dtetra, int * tetra_mem, 
		int ntetra, int subgridsize, int gridsize, int numsubgrid, float box){
	int loop_i = 0;
	int ind;
	float dx2 = box / gridsize / 2.0;
	ind = blockIdx.x * blockDim.x + threadIdx.x;		//the index of the tetrahedron
	if(ind >= numsubgrid){
		return;
	}
	tetra_mem[ind] = 0;
	int subsubgridsize = gridsize / subgridsize;

	Point v1, v8;
	v1 = getPoint(ind, 0, 0, 0,subgridsize, gridsize, subsubgridsize, box);
	v8 = getPoint(ind, subgridsize ,subgridsize, subgridsize, subgridsize, gridsize, subsubgridsize, box);

	for(loop_i = 0; loop_i < ntetra; loop_i ++){
		Tetrahedron * tetra = &(dtetra[loop_i]);
		//check whether the tetra is getting in touch with the current tetra
		if(isInTouch(ind, subgridsize, gridsize, subsubgridsize, box, dx2, tetra, v1, v8)){
			//if(loop_i == 3165)
			tetra_mem[ind] += 1;
		}
	}
}

//compute the actual list of tetrahedrons thar are in touch with subblock
__global__ void computeTetraSelection(Tetrahedron * dtetra, int * tetra_mem, int * tetra_select, 
		int ntetra, int subgridsize, int gridsize, int numsubgrid, float box,
		int start_ind, int end_ind){
	int loop_i = 0;
	int ind;
	float dx2 = box / gridsize / 2.0;
	ind = blockIdx.x * blockDim.x + threadIdx.x;		//the index of the tetrahedron

	if(ind < start_ind || ind >= end_ind){
		return;
	}

	if(ind >= numsubgrid){
		return;
	}
	int count = 0;
	int startind = 0;
	if(ind > 0){
		startind = tetra_mem[ind - 1];
	}

	int subsubgridsize = gridsize / subgridsize;
	int total = tetra_mem[ind] - startind;

	Point v1, v8;
	v1 = getPoint(ind, 0, 0, 0,subgridsize, gridsize, subsubgridsize, box);
	v8 = getPoint(ind, subgridsize ,subgridsize, subgridsize, subgridsize, gridsize, subsubgridsize, box);

	for(loop_i = 0; (loop_i < ntetra) && (count < total); loop_i ++){
		Tetrahedron * tetra = &dtetra[loop_i];
		//check whether the tetra is getting in touch with the current tetra
		if(isInTouch(ind, subgridsize, gridsize, subsubgridsize, box, dx2, tetra, v1, v8)){
			//if(loop_i == 3165)
			tetra_select[startind + count] = loop_i;
			count = count + 1;
		}
	}
}

//initialize the CUDA
hipError_t initialCUDA(TetraStream * tetrastream_, GridManager * gridmanager_, int mem_for_tetralist, GridVelocityManager * gridvelocity_, bool isVelocity_){
	is_Velocity = isVelocity_;
	grid_velocity = gridvelocity_;

	//int grid_size;
	TETRA_LIST_MEM_LIM = mem_for_tetralist;

	tetrastream = tetrastream_;
	gridmanager = gridmanager_;
	//tetras_v = tetrastream_->getTretras();

	num_tetra_ = tetrastream->getBlockSize();
	num_tetra_ = 6 * num_tetra_ * num_tetra_ * num_tetra_ * 8;

	sub_grid_size_ = gridmanager->getSubGridSize();
	//grid_size = gridmanager->getGridSize();

	//printf("%d\n", grid_size);
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

	// Allocate GPU buffers for tetrahedrons    .
    cudaStatus = hipMalloc((void**)&dev_tetras, num_tetra_ * sizeof(Tetrahedron));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating tetra memory!");
        return cudaStatus;
    }

	// Allocate GPU buffers for grids.
    cudaStatus = hipMalloc((void**)&dev_grids, sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(REAL));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating grids memory!");
        return cudaStatus;
    }

	if(is_Velocity){
		// Allocate GPU buffers for grids velocity.
		cudaStatus = hipMalloc((void**)&dev_grid_velocity, sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(Point));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed -- allocating grids memory for velocity!");
			return cudaStatus;
		}
	}

	// Allocate GPU tetra memory for subgrids.
	int nsub = gridmanager->getSubGridNum();

	cudaStatus = hipMalloc((void**)&dev_tetra_mem, nsub * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating grids memory!");
        return cudaStatus;
    }
	return cudaStatus;
}


//compute how many tetrahedrons are in touch with a certain subblock of the density grid
hipError_t computeTetraMemWithCuda(){
	//copy the memory to CUDA
	hipError_t cudaStatus;

	tetras_v = tetrastream ->getCurrentBlock();
	num_tetra_ = tetrastream->getBlockNumTetra();

	int blocksize = 512;
	int gridsize = gridmanager->getSubGridNum() / blocksize + 1;

	cudaStatus = hipMemcpy(dev_tetras, tetras_v, num_tetra_ * sizeof(Tetrahedron), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed -- copying tetrahedrons!\n");
        return cudaStatus;
    }

	//<<<gridsize, blocksize>>>
	computeTetraMem<<<gridsize, blocksize>>>(dev_tetras, dev_tetra_mem, 
		num_tetra_, gridmanager->getSubGridSize(), gridmanager->getGridSize(), 
		gridmanager->getSubGridNum(), 
		gridmanager->getEndPoint().x - gridmanager->getStartPoint().x);

	cudaStatus = hipDeviceSynchronize();
	if( cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize error -- sync tetra mem: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return cudaStatus;
}

//compute the tetraselection
//if has more grid need to be calculate, assign hasmore to be true, otherwise to be false
hipError_t computeTetraSelectionWithCuda(bool & hasmore){
	int blocksize = 512;
	int gridsize = gridmanager->getSubGridNum() / blocksize + 1;
	hipError_t cudaStatus;
	int memoryneed = 0;
	int start_index_tetra = current_tetra_list_ind;
	if(current_tetra_list_ind == 0){
		tetramem = new int[gridmanager->getSubGridNum()];
		tetramem_list = new int[gridmanager->getSubGridNum()];
		cudaStatus = hipMemcpy(tetramem_list, dev_tetra_mem, gridmanager->getSubGridNum() * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed -- copying tetra-mem!\n");
			return cudaStatus;
		}
		int j;
		//total_tetra_list_count = gridmanager->getSubGridNum();
		//tetramem_list[gridmanager->getSubGridNum()-1] = tetramem[gridmanager->getSubGridNum()-1];
		tetramem[0] = tetramem_list[0];
		for(j = 1; j < gridmanager->getSubGridNum(); j++){
			if(memoryneed ==0){
				tetramem[j] = tetramem_list[j] + tetramem[j - 1];
				//test
				//printf("%d\n",  tetramem_list[j]);

				if(tetramem[j] * 4 > TETRA_LIST_MEM_LIM){
					memoryneed = tetramem[j - 1];
					tetramem[j] = memoryneed;
					current_tetra_list_ind = j;
				}
			}else{
				tetramem[j] = memoryneed;
			}
		}

		if(memoryneed == 0){
			memoryneed = tetramem[gridmanager->getSubGridNum()-1];
			current_tetra_list_ind = gridmanager->getSubGridNum();
		}
	}else{
		int j;
		for(j = 0; j < current_tetra_list_ind; j++){
			tetramem[j] = 0;
		}
		//at least count 1
		tetramem[current_tetra_list_ind] = tetramem_list[current_tetra_list_ind];
		current_tetra_list_ind ++;
		
		for(j = current_tetra_list_ind; j < gridmanager->getSubGridNum(); j++){
			if(memoryneed ==0){

				//test
				//printf("%d\n",  tetramem_list[j]);

				tetramem[j] = tetramem_list[j] + tetramem[j - 1];
				if(tetramem[j] * 4 > TETRA_LIST_MEM_LIM){
					memoryneed = tetramem[j - 1];
					tetramem[j] = memoryneed;
					current_tetra_list_ind = j;
				}
			}else{
				tetramem[j] = memoryneed;
			}
		}
		if(memoryneed == 0){
			memoryneed = tetramem[gridmanager->getSubGridNum()-1];
			current_tetra_list_ind = gridmanager->getSubGridNum();
		}
	}

	//test
	/*int tttmm[512];
	int tttmm1[512];
	for(int i = 0; i < 512; i++){
		tttmm[i] = tetramem[i];
		tttmm1[i] = tetramem_list[i];
	}*/

	cudaStatus = hipMemcpy(dev_tetra_mem, tetramem, gridmanager->getSubGridNum() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed -- copying tetra-mem back!\n");
        return cudaStatus;
    }

	//allocating memory
	//printf("Tetramem: %d\n", tetramem[ gridmanager->getSubGridNum() - 1]);
	int totalmem = memoryneed;

	//test
	//testmemtttt += memoryneed;

	//printf("Memory allocating: %d\n", totalmem);
	hipFree(dev_tetra_select);
	cudaStatus = hipMalloc((void**)&dev_tetra_select, totalmem * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed -- allocating tetra memory!\n");
        return cudaStatus;
    }

	computeTetraSelection<<<gridsize, blocksize>>>(dev_tetras, dev_tetra_mem, dev_tetra_select, 
		num_tetra_, gridmanager->getSubGridSize(), gridmanager->getGridSize(), 
		gridmanager->getSubGridNum(), 
		gridmanager->getEndPoint().x - gridmanager->getStartPoint().x,
		start_index_tetra, current_tetra_list_ind);

	cudaStatus = hipDeviceSynchronize();
	if( cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize error -- sync tetra mem: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	//delete tetramem;
	hasmore = true;
	if(current_tetra_list_ind == gridmanager->getSubGridNum()){
		current_tetra_list_ind = 0;
		hasmore = false;
		delete tetramem;
		delete tetramem_list;
	}
	return hipSuccess;
}

//density estimation
hipError_t calculateGridWithCuda(){
	hipError_t cudaStatus;
	//dim3 size(sub_grid_size_, sub_grid_size_, sub_grid_size_);
	dim3 blocksize(8, 8, 8);
	dim3 gridsize(sub_grid_size_/8, sub_grid_size_/8, sub_grid_size_/8);
	if(sub_grid_size_ %8 != 0){
		gridsize.x ++;
		gridsize.y ++;
		gridsize.z ++;
	}

	if(num_tetra_ == 0)
		return hipErrorUnknown;

	cudaStatus = hipMemcpy(dev_grids, gridmanager->getSubGrid(), sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(REAL), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed -- copying subgrids!\n");
        return cudaStatus;
    }


	if(is_Velocity){
		cudaStatus = hipMemcpy(dev_grid_velocity, grid_velocity->getSubGrid(), sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(Point), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed -- copying subgrids of velocity!\n");
			return cudaStatus;
		}
	}

	//<<<1, size>>>
	Point p0 = gridmanager->getPoint(0,0,0);
	//<<<gridsize, blocksize>>>
	tetraSplatter<<<gridsize, blocksize>>>(dev_tetras, num_tetra_, dev_grids, gridmanager->getGridSize(), gridmanager->getSubGridSize(),
		dev_tetra_mem, dev_tetra_select, gridmanager->getCurrentInd(), gridmanager->getSubGridNum(),
		gridmanager->getEndPoint().x - gridmanager->getStartPoint().x, p0.x, p0.y, p0.z, is_Velocity, dev_grid_velocity);

	cudaStatus = hipDeviceSynchronize();
	if( cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize error -- estimate density: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(gridmanager->getSubGrid(), dev_grids, sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(REAL), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed -- copying subgrids!\n");
        return cudaStatus;
    }

	if(is_Velocity){
		cudaStatus = hipMemcpy(grid_velocity->getSubGrid(), dev_grid_velocity, sub_grid_size_ * sub_grid_size_ * sub_grid_size_ * sizeof(Point), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed -- copying subgrids of velocity back!\n");
			return cudaStatus;
		}
	}

	return hipSuccess;
}

//clean up
void finishCUDA(){
	hipFree(dev_grids);
	hipFree(dev_tetras);
	hipFree(dev_tetra_mem);
	hipFree(dev_tetra_select);
	if(is_Velocity){
		hipFree(dev_grid_velocity);
	}
}
// Helper function for using CUDA to add vectors in parallel.


//assuming the start coordinates is (0,0,0)
