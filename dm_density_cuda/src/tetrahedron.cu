#include "hip/hip_runtime.h"
/*
 * tetrahedron.cpp
 *
 *  Created on: Dec 20, 2012
 *      Author: lyang
 */
#include <cmath>
#include <algorithm>
#include <cstdio>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

#include "tetrahedron.h"

#define EPSILON 1e-6
#define EPSILON1 1e-11

CUDA_CALLABLE_MEMBER Tetrahedron::Tetrahedron(){
	volume = 0;
	minx_ = 0;
	miny_ = 0;
	minz_ = 0;
	maxx_ = 0;
	maxy_ = 0;
	maxz_ = 0;
}

CUDA_CALLABLE_MEMBER void Tetrahedron::computeMaxMin(){
	minx_ = min(min(min(v1.x, v2.x), v3.x), v4.x);
	miny_ = min(min(min(v1.y, v2.y), v3.y), v4.y);
	minz_ = min(min(min(v1.z, v2.z), v3.z), v4.z);
	maxx_ = max(max(max(v1.x, v2.x), v3.x), v4.x);
	maxy_ = max(max(max(v1.y, v2.y), v3.y), v4.y);
	maxz_ = max(max(max(v1.z, v2.z), v3.z), v4.z);
}

CUDA_CALLABLE_MEMBER REAL Tetrahedron::computeVolume(){
	REAL vol;
	REAL v1x, v1y, v1z;
	REAL v2x, v2y, v2z;
	REAL v3x, v3y, v3z;

	v1x = v2.x - v1.x;
	v1y = v2.y - v1.y;
	v1z = v2.z - v1.z;

	v2x = v3.x - v1.x;
	v2y = v3.y - v1.y;
	v2z = v3.z - v1.z;

	v3x = v4.x - v1.x;
	v3y = v4.y - v1.y;
	v3z = v4.z - v1.z;

	vol =  v1x*v2y*v3z + v1y*v2z*v3x + v1z*v2x*v3y -
	      (v1z*v2y*v3x + v1y*v2x*v3z + v1x*v2z*v3y);
	vol /= 6.0;
	volume = abs(vol);

	//compute min and max
	computeMaxMin();
	
	//compute d0 to reduce calculation
	//double m[4][4];
	//c2m(v1, v2, v3, v4, m);		//change the det to be det / 10^11
	//d0 = det4d(m);

	return vol;
}

CUDA_CALLABLE_MEMBER double Tetrahedron::det4d(double m[4][4]) {
   double value;
   double v1, v2, v3, v4, v5, v6, v7, v8, v9, v10, v11, v12;
		 v1 =  (m[0][3] * m[1][2] * m[2][1] * m[3][0]-m[0][2] * m[1][3] * m[2][1] * m[3][0]);
		 v2 =  (-m[0][3] * m[1][1] * m[2][2] * m[3][0]+m[0][1] * m[1][3] * m[2][2] * m[3][0]);
		 v3 =  (+m[0][2] * m[1][1] * m[2][3] * m[3][0]-m[0][1] * m[1][2] * m[2][3] * m[3][0]);
		 v4 =  (-m[0][3] * m[1][2] * m[2][0] * m[3][1]+m[0][2] * m[1][3] * m[2][0] * m[3][1]);
		 v5 =  (+m[0][3] * m[1][0] * m[2][2] * m[3][1]-m[0][0] * m[1][3] * m[2][2] * m[3][1]);
		 v6 =  (-m[0][2] * m[1][0] * m[2][3] * m[3][1]+m[0][0] * m[1][2] * m[2][3] * m[3][1]);
		 v7 =  (+m[0][3] * m[1][1] * m[2][0] * m[3][2]-m[0][1] * m[1][3] * m[2][0] * m[3][2]);
		 v8 =  (-m[0][3] * m[1][0] * m[2][1] * m[3][2]+m[0][0] * m[1][3] * m[2][1] * m[3][2]);
		 v9 =  (+m[0][1] * m[1][0] * m[2][3] * m[3][2]-m[0][0] * m[1][1] * m[2][3] * m[3][2]);
		 v10 = (-m[0][2] * m[1][1] * m[2][0] * m[3][3]+m[0][1] * m[1][2] * m[2][0] * m[3][3]);
		 v11 = (+m[0][2] * m[1][0] * m[2][1] * m[3][3]-m[0][0] * m[1][2] * m[2][1] * m[3][3]);
		 v12 = (-m[0][1] * m[1][0] * m[2][2] * m[3][3]+m[0][0] * m[1][1] * m[2][2] * m[3][3]);
   value = (v1 + v2 + v3 + v4 +  v5 + v6 + v7 + v8 + v9 + v10 + v11 + v12);
   return value;
}

CUDA_CALLABLE_MEMBER void Tetrahedron::c2m(Point p1, Point p2, Point p3, Point p4, double m[4][4]){
	m[0][0] = p1.x * 1.0e-11;
	m[0][1] = p1.y * 1.0e-11;
	m[0][2] = p1.z * 1.0e-11;
	m[0][3] = 1.0 * 1.0e-11;
	m[1][0] = p2.x;
	m[1][1] = p2.y;
	m[1][2] = p2.z;
	m[1][3] = 1.0f;
	m[2][0] = p3.x;
	m[2][1] = p3.y;
	m[2][2] = p3.z;
	m[2][3] = 1.0f;
	m[3][0] = p4.x;
	m[3][1] = p4.y;
	m[3][2] = p4.z;
	m[3][3] = 1.0f;
}

CUDA_CALLABLE_MEMBER bool Tetrahedron::isInTetra(Point p){
	if(p.x > maxx() || p.y > maxy() || p.z > maxz()
	|| p.x < minx() || p.y < miny() || p.z < minz()){
		return false;
	}
	double m[4][4];
	double d0=0.0, d1=0.0, d2=0.0, d3=0.0, d4=0.0;

	c2m(v1, v2, v3, v4, m);		//change the det to be det / 10^11
	d0 = det4d(m);

	c2m(p, v2, v3, v4, m);
	d1 = det4d(m);

	c2m(v1, p, v3, v4, m);
	d2 = det4d(m);

	c2m(v1, v2, p, v4, m);
	d3 = det4d(m);

	c2m(v1, v2, v3, p, m);
	d4 = det4d(m);


	if(d0 > 0){
		return (d1 >= 0) && (d2 >= 0) && (d3 >= 0) && (d4 >= 0);
	}else{
		return (d1 <= 0) && (d2 <= 0) && (d3 <= 0) && (d4 <= 0);
	}
}


CUDA_CALLABLE_MEMBER REAL Tetrahedron::minx(){
	return minx_;
}
CUDA_CALLABLE_MEMBER REAL Tetrahedron::miny(){
	return miny_;
}
CUDA_CALLABLE_MEMBER REAL Tetrahedron::minz(){
	return minz_;
}
CUDA_CALLABLE_MEMBER REAL Tetrahedron::maxx(){
	return maxx_;
}
CUDA_CALLABLE_MEMBER REAL Tetrahedron::maxy(){
	return maxy_;
}
CUDA_CALLABLE_MEMBER REAL Tetrahedron::maxz(){
	return maxz_;
}


CUDA_CALLABLE_MEMBER Point &  Point::operator=(const Point &rhs){
	this->x = rhs.x;
	this->y = rhs.y;
	this->z = rhs.z;
	return *this;
}

CUDA_CALLABLE_MEMBER Point::Point(const Point &point){
	this->x = point.x;
	this->y = point.y;
	this->z = point.z;
}


CUDA_CALLABLE_MEMBER Tetrahedron & Tetrahedron::operator=(const Tetrahedron & rhs){
	this->v1 = rhs.v1;
	this->v2 = rhs.v2;
	this->v3 = rhs.v3;
	this->v4 = rhs.v4;
	this->volume = rhs.volume;
	this->maxx_ = rhs.maxx_;
	this->maxy_ = rhs.maxy_;
	this->maxz_ = rhs.maxz_;
	this->minx_ = rhs.minx_;
	this->miny_ = rhs.miny_;
	this->minz_ = rhs.minz_;
	return *this;
}

CUDA_CALLABLE_MEMBER Tetrahedron::Tetrahedron(const Tetrahedron & rhs){
	this->v1 = rhs.v1;
	this->v2 = rhs.v2;
	this->v3 = rhs.v3;
	this->v4 = rhs.v4;
	this->volume = rhs.volume;
	this->maxx_ = rhs.maxx_;
	this->maxy_ = rhs.maxy_;
	this->maxz_ = rhs.maxz_;
	this->minx_ = rhs.minx_;
	this->miny_ = rhs.miny_;
	this->minz_ = rhs.minz_;
}

CUDA_CALLABLE_MEMBER Point::Point(){
	x = 0;
	y = 0;
	z = 0;
}